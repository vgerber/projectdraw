#include "hip/hip_runtime.h"
#include "gol.h"
#include <stdio.h>
#include <iostream>


#define HANDLE_ERROR(err) (handleCudaError(err, __FILE__, __LINE__))

const char* hipGetErrorString(hipError_t error);


const int DEVICE_ID = 0;
int DEVICE_SM = 0;


static void handleCudaError(hipError_t err, const char *file, int line) {
	if (err != hipSuccess) {
		printf("CUDA Error %s in %s %d\n", hipGetErrorString(err), file, line);
		exit(err);
	}
}

__global__ void testp(int N) {
	for (int id = blockIdx.x * blockDim.x + threadIdx.x; id < N; id += blockDim.x * gridDim.x) {
		printf("Test #%d\n", id);
	}
}

__global__ void GOL_Ghost_GPU(int dim, int *world) {
	int i, id;

	//striding loop
	for (id = blockIdx.x * blockDim.x + threadIdx.x; id <= dim + 1; id += blockDim.x * gridDim.x) {
		//row
		//i = id / (dim + 2);
		//col
		//j = id % (dim + 2);

		i = id;
		if (i >= 1 && i <= dim) {
			//TODO: Copy first real row to bottom ghost row
			world[(dim + 1) * (dim + 2) + i] = world[(dim + 2) + i];
			//TODO: Copy last real row to top ghost row
			world[i] = world[dim * (dim + 2) + i];
		}

		//TODO: Copy first real column to right most ghost column
		world[i * (dim + 2) + (dim + 1)] = world[i * (dim + 2) + 1];
		//TODO: Copy last real column to left most ghost column
		world[i * (dim + 2)] = world[i * (dim + 2) + dim];


	}
}

__global__ void GOL_GPU(int dim, int *world, int *newWorld, uchar4 * texture) {
	int id;

	for (id = blockIdx.x * blockDim.x + threadIdx.x; id < (dim + 2) * (dim + 2); id += blockDim.x * gridDim.x) {
		int x = id % (dim + 2);
		if (x == 0 || x == (dim + 1)) {
			texture[id].x = 100;
			texture[id].y = 210;
			texture[id].z = 100;
			continue;
		}
		int y = floorf((float)id / (float)(dim + 2));
		if(y == 0 || y == (dim + 1)) {
			texture[id].x = 100;
			texture[id].y = 210;
			texture[id].z = 100;
			continue;
		}
		//world point
		int cell = world[id];
		int numNeighbors;

		// Get the number of neighbors for a world point
		numNeighbors = world[id + (dim + 2)]   //TODO: lower
			+ world[id - (dim + 2)] //TODO: upper
			+ world[id + 1]         //TODO: right
			+ world[id - 1]         //TODO: left

			+ world[id + (dim + 3)]  //TODO: diagonal lower right
			+ world[id - (dim + 1)]  //TODO: diagonal upper right
			+ world[id + (dim + 1)]  //TODO: diagonal lower left
			+ world[id - (dim + 3)]; //TODO: diagonal upper leftDivision Mo

// game rules for Conways 23/3-world
// 1) Any live cell with fewer than two live neighbours dies
		if (cell == 1 && numNeighbors < 2) //TODO
			newWorld[id] = 0;

		// 2) Any live cell with two or three live neighbours lives
		else if (cell == 1 && numNeighbors == 2 || numNeighbors == 3) //TODO
			newWorld[id] = 1;

		// 3) Any live cell with more than three live neighbours dies
		else if (cell == 1 && numNeighbors > 3) //TODO
			newWorld[id] = 0;

		// 4) Any dead cell with exactly three live neighbours becomes a live cell
		else if (cell == 0 && numNeighbors == 3) //TODO
			newWorld[id] = 1;

		else
			newWorld[id] = cell;
		int value = newWorld[id];
		texture[id].x = value ? 255 : 0;
		texture[id].y = value ? 255 : 0;
		texture[id].z = value ? 255 : 0;
	}
}

__global__ void GOL_GPU_CPY(int dim, int *world, int *newWorld, uchar4 * texture) {
	int id;

	for (id = blockIdx.x * blockDim.x + threadIdx.x; id < (dim + 2) * (dim + 2); id += blockDim.x * gridDim.x) {
		int x = id % (dim + 2);
		if (x == 0 || x == (dim + 1)) {
			texture[id].x = 100;
			texture[id].y = 210;
			texture[id].z = 100;
			continue;
		}
		int y = floorf((float)id / (float)(dim + 2));
		if (y == 0 || y == (dim + 1)) {
			texture[id].x = 100;
			texture[id].y = 210;
			texture[id].z = 100;
			continue;
		}
		//world point
		int numNeighbors;

		// Get the number of neighbors for a world point
		numNeighbors = world[id + (dim + 2)]   //TODO: lower
			+ world[id - (dim + 2)] //TODO: upper
			+ world[id + 1]         //TODO: right
			+ world[id - 1]         //TODO: left

			+ world[id + (dim + 3)]  //TODO: diagonal lower right
			+ world[id - (dim + 1)]  //TODO: diagonal upper right
			+ world[id + (dim + 1)]  //TODO: diagonal lower left
			+ world[id - (dim + 3)]; //TODO: diagonal upper leftDivision Mo

		int cell = numNeighbors % 2;

		newWorld[id] = cell;

		texture[id].x = cell ? 255 : 0;
		texture[id].y = cell ? 255 : 0;
		texture[id].z = cell ? 255 : 0;
	}
}

void initGlider(int dim , int *world) {
	for(int i = 1; i <= dim; i++) {
	  for(int j = 1; j <= dim; j++) {
		int value = 0;
		if(i == 2 && j == 3) {
		  value = 1;
		}
		if(i == 3 && j == 4) {
		  value = 1;
		}      
		if(i == 4 && j >= 2 && j <= 4) {
		  value = 1;
		}
		world[i * (dim + 2) + j] = value;
	  }
	}
  }
  
  void initOscillator(int dim , int *world) {
	for(int i = 1; i <= dim; i++) {
	  for(int j = 1; j <= dim; j++) {
		int value = 0;
		if(i == 3 && j == 3) {
		  value = 1;
		}
		if(i == 3 && j == 4) {
		  value = 1;
		}      
		if(i == 4 && j == 3) {
		  value = 1;
		}
		if(i == 5 && j == 6) {
		  value = 1;
		}
		if(i == 6 && j == 5) {
		  value = 1;
		}
		if(i == 6 && j == 6) {
		  value = 1;
		}
		world[i * (dim + 2) + j] = value;
	  }
	}
  }
  
  void initStatic(int dim , int *world) {
	for(int i = 1; i <= dim; i++) {
	  for(int j = 1; j <= dim; j++) {
		int value = 0;
		if(i >= 4 && i <= 5) {
		  if(j >= 4 && j <= 5) {
			value = 1;
		  }
		}
		world[i * (dim + 2) + j] = value;
	  }
	}
  }

  void initH(int dim, int *world) {
	  for (int i = 1; i <= dim; i++) {
		  for (int j = 1; j <= dim; j++) {
			  int value = 0;
			  if (i == 512 ||i == 513) {
				  if (j == 512 || j == 513)
				  {
					  value = 1;
				  }
			  }
			  if (i >= 510 && i <= 515) {
				  if (j == 511 || j == 514) {
					  value = 1;
				  }
			  }
			  world[i * (dim + 2) + j] = value;
		  }
	  }
  }

  void initRandom(int dim, int *world)
  {
  
	int i, j;
  
	// Assign initial population randomly
	srand(time(NULL));
	for (i = 1; i <= dim; i++)
	{
	  for (j = 1; j <= dim; j++)
	  {
		world[i * (dim + 2) + j] = rand() % 2;
	  }
	}
  }

__global__ void setTextureData(int dim,  int * world, uchar4 * texture) {
	for (int id = blockIdx.x * blockDim.x + threadIdx.x; id < (dim + 2) * (dim + 2); id += blockDim.x * gridDim.x) {
		int value = world[id];
		texture[id].x = value ? 255 : 0;
		texture[id].y = value ? 255 : 0;
		texture[id].z = value ? 255 : 0;	
	}
} 


void initGOL(int ** h_world, int ** d_world, int ** d_tmpWorld, int dim, uchar4 * texture) {
	hipDeviceGetAttribute(&DEVICE_SM, hipDeviceAttributeMultiprocessorCount, DEVICE_ID);
	printf("SMs: %d\n", DEVICE_SM);

	size_t worldBytes = (dim + 2) * (dim + 2) * sizeof(int);
	*h_world = (int*)malloc(worldBytes);
	HANDLE_ERROR(hipMalloc(d_world, worldBytes));
	HANDLE_ERROR(hipMalloc(d_tmpWorld, worldBytes));

	//initGlider(dim, *h_world);
	//initRandom(dim, *h_world);
	initH(dim, *h_world);

	HANDLE_ERROR(hipMemcpy(*d_world, *h_world, worldBytes, hipMemcpyHostToDevice));

	setTextureData<<<32*DEVICE_SM, 256>>>(dim, *d_world, texture);
	hipDeviceSynchronize();
	HANDLE_ERROR(hipGetLastError());
}

void stepGOL(int * h_world, int * d_world, int * d_tmpWorld, int dim, uchar4 * texture) {
	size_t worldBytes = (dim + 2) * (dim + 2) * sizeof(int);
	GOL_Ghost_GPU<<<32*DEVICE_SM, 256>>>(dim, d_world);
	hipDeviceSynchronize();
    HANDLE_ERROR(hipGetLastError());
    
	//main world
	//GOL_GPU<<<32*DEVICE_SM, 256>>>(dim, d_world, d_tmpWorld, texture);	
	GOL_GPU_CPY<<<32*DEVICE_SM, 256>>>(dim, d_world, d_tmpWorld, texture);	
	hipDeviceSynchronize();
	HANDLE_ERROR(hipGetLastError());
    //world swap
    HANDLE_ERROR(hipMemcpy(d_world, d_tmpWorld, worldBytes, hipMemcpyDeviceToDevice)); 
	HANDLE_ERROR(hipMemcpy(h_world, d_tmpWorld, worldBytes, hipMemcpyDeviceToHost)); 
}

void testprint() {

	int devId = 0;
	int gpuSMs;
	hipDeviceGetAttribute(&gpuSMs, hipDeviceAttributeMultiprocessorCount, devId);
	printf("SMs: %d\n", gpuSMs);

	testp<<<32 * gpuSMs, 256>>>(10);
	hipDeviceSynchronize();
	HANDLE_ERROR(hipGetLastError());
}